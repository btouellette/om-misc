#ifndef _PRESCAN_CU_
#define _PRESCAN_CU_

// includes, kernels
#include <assert.h>


#define NUM_BANKS 16
#define LOG_NUM_BANKS 4
// MP4 - You can use any other block size you wish.
#define BLOCK_SIZE 256

// MP4 - Host Helper Functions (allocate your own data structure...)


// MP4 - Device Functions


// MP4 - Kernel Functions


// **===-------- MP4 - Modify the body of this function -----------===**
// You may need to make multiple kernel calls, make your own kernel
// function in this file, and then call them from here.
void prescanArray(float *outArray, float *inArray, int numElements)
{



}
// **===-----------------------------------------------------------===**


#endif // _PRESCAN_CU_
