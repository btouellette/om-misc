#include "hip/hip_runtime.h"
#ifndef _2DCONVOLUTION_KERNEL_H_
#define _2DCONVOLUTION_KERNEL_H_

#include <stdio.h>
#include "2Dconvolution.h"

#define N_SIZE BLOCK_SIZE+KERNEL_SIZE-1
#define OFF KERNEL_SIZE/2

// Matrix multiplication kernel thread specification
__global__ void ConvolutionKernel(float *M, float *N, float *P, int M_h, int M_w, int N_h, int N_w)
{
	// For 5x5 kernel
	// C(i,j) = sum (m = 0 to 4) { sum(n = 0 to 4) { A[m][n] * B[i+m-2][j+n-2] } }
	// where 0 <= i < B.height and 0 <= j < B.width
	int P_h = N_h;
	int P_w = N_w;

	__shared__ float Mds[KERNEL_SIZE][KERNEL_SIZE];
	__shared__ float Nds[N_SIZE][N_SIZE];

	int bx = blockIdx.x;
	int by = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int row = by*BLOCK_SIZE + ty;
	int col = bx*BLOCK_SIZE + tx;

	float Pvalue = 0.0;

	// Load in the kernel using a tiled approach
	for(int i = 0; i <= KERNEL_SIZE/BLOCK_SIZE; i++)
	{
		for(int j = 0; j <= KERNEL_SIZE/BLOCK_SIZE; j++)
		{
			// Check that we are loading an address inside the kernel and then load it into shared memory
			if(tx+i*BLOCK_SIZE < KERNEL_SIZE &&
			   ty+j*BLOCK_SIZE < KERNEL_SIZE)
			{
				Mds[ty+j*BLOCK_SIZE][tx+i*BLOCK_SIZE] = M[(ty+j*BLOCK_SIZE)*M_w + tx+i*BLOCK_SIZE];
			}
		}
	}

	// Load in KERNEL_SIZE/2 around the block using a tiled approach
	for(int i = 1; i <= (KERNEL_SIZE/2)/BLOCK_SIZE; i++)
	{
		for(int j = 1; j <= (KERNEL_SIZE/2)/BLOCK_SIZE; j++)
		{
			int xds = tx+i*BLOCK_SIZE+OFF;
			int yds = ty+j*BLOCK_SIZE+OFF;
			// First check that the index we want is a valid element of N, then check that it is needed
			// It will be needed if it fits into our Nds which is sized for BLOCK_SIZE and KERNEL_SIZE/2 on either side
			if(xds < N_SIZE && yds < N_SIZE)
			{
				int x = col+i*BLOCK_SIZE;
				int y = row+j*BLOCK_SIZE;
				if(x < N_w && y < N_h)
				{
					// Load in the index
					Nds[yds][xds] = N[y*N_w + x];
				}
				else
				{
					Nds[yds][xds] = 0.0;
				}
			}
		}
	}

	// Don't do anything if we aren't operating on a valid pixel
	if(row < P_h && col < P_w)
	{
		// Load in entire block to shared memory
		Nds[ty+OFF][tx+OFF] = N[row*N_w + col];
		// Ensure all threads have access to the shared memory loads
		__syncthreads();

		unsigned int m_b = (row < OFF)? OFF - row : 0;
		unsigned int m_e = (row >= (N_h - OFF))? N_h - row + OFF : KERNEL_SIZE;
		unsigned int n_b = (col < OFF)? OFF - col : 0;
		unsigned int n_e = (col >= (N_w - OFF))? N_w - col + OFF : KERNEL_SIZE;
		for(int m = m_b; m < m_e; m++)
		{
			for(int n = n_b; n < n_e; n++)
			{
				Pvalue += Mds[m][n]*N[(m+row-OFF)*N_w + n+col-OFF];
				//Pvalue += Mds[m][n]*Nds[m+ty][n+tx];
			}
		}
		P[row*P_w + col] = Pvalue;
	}
}

#endif // #ifndef _2DCONVOLUTION_KERNEL_H_
