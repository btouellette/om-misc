#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2006 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and 
 * international Copyright laws.  
 *
 * This software and the information contained herein is PROPRIETARY and 
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and 
 * conditions of a Non-Disclosure Agreement.  Any reproduction or 
 * disclosure to any third party without the express written consent of 
 * NVIDIA is prohibited.     
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE 
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH 
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF 
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.   
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL, 
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS 
 * OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE 
 * OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE 
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.  This source code is a "commercial item" as 
 * that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of 
 * "commercial computer software" and "commercial computer software 
 * documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995) 
 * and is provided to the U.S. Government only as a commercial end item.  
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through 
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

#ifndef _2DCONVOLUTION_KERNEL_H_
#define _2DCONVOLUTION_KERNEL_H_

#include <stdio.h>
#include "2Dconvolution.h"

// Matrix multiplication kernel thread specification
__global__ void ConvolutionKernel(float *M, float *N, float *P, int M_h, int M_w, int N_h, int N_w)
{
	// For 5x5 kernel
	// C(i,j) = sum (m = 0 to 4) { sum(n = 0 to 4) { A[m][n] * B[i+m-2][j+n-2] } }
	// where 0 <= i < B.height and 0 <= j < B.width
	int P_h = N_h;
	int P_w = N_w;

	__shared__ float Mds[KERNEL_SIZE][KERNEL_SIZE];
	__shared__ float Nds[BLOCK_SIZE+KERNEL_SIZE-1][BLOCK_SIZE+KERNEL_SIZE-1];

	int bx = blockIdx.x;
	int by = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int row = by*BLOCK_SIZE + ty;
	int col = bx*BLOCK_SIZE + tx;
	int off = KERNEL_SIZE/2;

	float Pvalue = 0.0;

	// Don't do anything if we aren't operating on a valid pixel
	if(row < P_h && col < P_w)
	{
		// Load in the kernel. Must satisfy KERNEL_SIZE < BLOCK SIZE
		if(tx < KERNEL_SIZE && ty < KERNEL_SIZE)
			Mds[ty][tx] = M[ty*M_w + tx];
		
		// Load in the entire block to shared memory
		Nds[ty][tx] = N[row*N_w + col];
		// Still need KERNEL_SIZE/2 on either side for convolution
		if(tx == 0)
		{
			for(int i = 0; i <= off; i++)
				Nds[ty+off][off-i] = N[row*N_w + (col-i)];
		}
		else if(tx == BLOCK_SIZE-1)
		{
			for(int i = 0; i <= off; i++)
				Nds[ty+off][tx+off+i] = N[row*N_w + (col+i)];
		}
		if(ty == 0)
		{
			for(int i = 0; i <= off; i++)
				Nds[off-i][tx+off] = N[(row-i)*N_w + col];
		}
		else if(ty == BLOCK_SIZE-1)
		{
			for(int i = 0; i <= off; i++)
				Nds[ty+off+i][tx+off] = N[(row+i)*N_w + col];
		}
		__syncthreads();

		unsigned int m_b = (row < 2)? 2 - row : 0;
		unsigned int m_e = (row > (N_h - 3))? N_h - row + 2 : 5;
		unsigned int n_b = (col < 2)? 2 - col : 0;
		unsigned int n_e = (col > (N_w - 3))? N_w - col + 2 : 5;
		for(int m = m_b; m < m_e; m++)
		{
			for(int n = n_b; n < n_e; n++)
			{
				Pvalue += Mds[m][n]*N[(m+row-2)*N_w + n+col-2];
			}
		}
		P[row*P_w + col] = Pvalue;
	}
}

#endif // #ifndef _2DCONVOLUTION_KERNEL_H_
