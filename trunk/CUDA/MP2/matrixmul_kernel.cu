#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2006 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and 
 * international Copyright laws.  
 *
 * This software and the information contained herein is PROPRIETARY and 
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and 
 * conditions of a Non-Disclosure Agreement.  Any reproduction or 
 * disclosure to any third party without the express written consent of 
 * NVIDIA is prohibited.     
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE 
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH 
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF 
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.   
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL, 
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS 
 * OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE 
 * OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE 
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.  This source code is a "commercial item" as 
 * that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of 
 * "commercial computer software" and "commercial computer software 
 * documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995) 
 * and is provided to the U.S. Government only as a commercial end item.  
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through 
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

/* Matrix multiplication: C = A * B.
 * Device code.
 */

#ifndef _MATRIXMUL_KERNEL_H_
#define _MATRIXMUL_KERNEL_H_

#include <stdio.h>
#include "matrixmul.h"

// Tile size has to be less than sqrt(512) == 23 since we can only have 512 threads in a block
#define TILE 16 

////////////////////////////////////////////////////////////////////////////////
//! Simple test kernel for device functionality
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
// Matrix multiplication kernel thread specification
__global__ void MatrixMulKernel(float* M, float* N, float* P, int M_h, int M_w, int N_w)
{
	int N_h = M_w;
	int P_h = M_h;
	int P_w = N_w;
	 
	__shared__ float Mds[TILE][TILE];
	__shared__ float Nds[TILE][TILE];

	int bx = blockIdx.x;
	int by = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int row = by*TILE + ty;
	int col = bx*TILE + tx;

	float Pvalue = 0.0;
	
	// For each tile
	int i;
	for(i = 0; i < M_w - TILE; i += TILE)
	{
		// Help load M and N tiles into shared memory
		Mds[ty][tx] = M[row*M_w + (i+tx)];
		Nds[ty][tx] = N[(i+ty)*N_w + col];
		// Ensure that every element is loaded
		__syncthreads();
		// Calculate this threads value
		for(int k = 0; k < TILE; ++k)
			Pvalue += Mds[ty][k]*Nds[k][tx];
		// Sync here to make sure that everyone is done using Mds and Nds
		__syncthreads();
	}
	
	// We still have to clean up the edges in case the matrix isn't aligned to tile size
	// Load in the value from M into the tile (or 0 if we are outside the matrix bounds)
	int index = row*M_w + (i+tx);
	if(index < M_h*M_w)
		Mds[ty][tx] = M[index];
	else
		Mds[ty][tx] = 0.0;

	// Load in the value from N into the tile (or 0 if we are outside the matrix bounds)
	index = (i+ty)*N_w + col;
	if(index < N_h*N_w)
		Nds[ty][tx] = N[index];
	else
		Nds[ty][tx] = 0.0;
	
	// Ensure that every element is loaded
	__syncthreads();
	for(int k = 0; k < TILE; ++k)
		Pvalue += Mds[ty][k]*Nds[k][tx];
		
	// Copy the element we calculated back if it is a valid element of P
	if(row < P_h && col < P_w)
		P[row*P_w + col] = Pvalue;
}

#endif // #ifndef _MATRIXMUL_KERNEL_H_
